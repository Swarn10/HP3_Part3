#include "hip/hip_runtime.h"
#include "im2col.hpp"
#define SHMEM_SIZE 32
__global__ void matrixMul(const float *A,const float *B, float *C, int m, int n, int k, int batch_size) {
    // Compute each thread's global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Statically allocated shared memory
    __shared__ float s_a[SHMEM_SIZE];
    __shared__ float s_b[SHMEM_SIZE];
    for(int t = 0;t < batch_size; t++) {
        // Accumulate in temporary variable
        float tmp = 0;
        // Sweep tile across matrix
        for (int i = 0; i < k; i += blockDim.x) {
          // Load in elements for this tile
          s_a[threadIdx.y * blockDim.x + threadIdx.x] = A[row * k + i + threadIdx.x + t*m*k];
          s_b[threadIdx.y * blockDim.x + threadIdx.x] = B[i * n + threadIdx.y * n + col];

          // Wait for both tiles to be loaded in before doing computation
          __syncthreads();

          // Do matrix multiplication on the small matrix
          for (int j = 0; j < blockDim.x; j++) {
            tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
          }

          // Wait for all threads to finish using current tiles before loading in new
          // ones
          __syncthreads();
    }
    
    // Write back results
    C[row * n + col +  t*m*n] = tmp;
  }  
}

// converts a batch of images of shape: data_im: batch x ic x ih x iw (ic: input_channels in image)
// to 2D col of shape: data_col: batch x (ic * kh * kw) x (hcol * wcol)
// filter size: kh x kw
// kernel multiplication patches: hcol x wcol (Based on input size, kernel size, padding, stride)
// Each thread writes one kernel multiplication patch (kh x kw) in data_col
// n is the number of tasks (here: ic * hcol * wcol, ie number of kernel patches per image)
__global__ void im2col_kernel(const float * data_im, float * data_col, const int n,
							  const int kh, const int kw, const int pad, const int stride,
							  const int ih, const int iw, const int ic,
							  const int hcol, const int wcol) 
{
	// esentially this loop could have run batch size number of times
	// but since we are launching enough threads to handle each image separately, it executes just once
	// here it is majorly prevents any extra threads we launch from accessing memory
	CUDA_KERNEL_LOOP(index, n)
	{
		// figure out which part of which image you will work on
		int imidx = blockIdx.y;
		int w_out = index % wcol;
		index /= wcol;
		int h_out = index % hcol;
		int channel_in = index / hcol;
		int h_in = h_out * stride - pad;
		int w_in = w_out * stride - pad;
		// this thread will write the output patch (kh x kw) at location (imidx, channel_out, h_out, w_out)
		// that patch is based on the image patch at (imidx, channel_in, h_in, w_in)
		// i.e. will do the work for patch centred at (channel_in, h_in, w_in) in image imidx
		data_im += ((imidx * ic + channel_in) * ih + h_in) * iw + w_in;
		data_col += ((imidx * ic + channel_in) * kh * kw * hcol + h_out) * wcol + w_out;
		#pragma unroll
		for (int i = 0; i < kh; ++i) {
			for (int j = 0; j < kw; ++j) {
				int h = h_in + i;
				int w = w_in + j;
				*data_col = (h >= 0 && w >= 0 && h < ih && w < iw) ?
				  data_im[i * iw + j]: 0;
				data_col += hcol * wcol;
			}
		}
	}
}

// takes a batch of images on GPU: bs x ic x ih x iw (ic: input channels, bs: batch size)
// and the kernels on GPU: oc x ic x kh x kw (oc: output channels)
// does the convolution based on padding (pad) and stride
// data_col is used for intermediate col form storage
// output is returned in data_out
void im2col_gemm_gpu(const float * data_im, const float * data_ker,
					 const int kh, const int kw, const int pad, const int stride,
					 const int ih, const int iw, const int ic, const int oc,
					 float * data_col, float * data_out, int bs)
{
	//hipDeviceProp_t devp;
	//hipGetDeviceProperties(&devp, 0);
	//unsigned int SHMEM_SIZE = devp.sharedMemPerBlock/(2*sizeof(float));
	//printf("Total shared memory per block:%u\n", SHMEM_SIZE);
	// Step 1: convert the image to col form
	
	// dimensions of the col corr to this image
	int hcol = (ih + 2 * pad - kh) / stride + 1;
	int wcol = (iw + 2 * pad - kw) / stride + 1;

	// We are going to launch bs groups of ic * hcol * wcol kernels threads for im2col,
	// each thread is responsible for copying a single-channel kernel multiplication patch
	// i.e. one thread per output pixel in the output of conv
	// So, all images in batch are converted to col form parallely
	int op_size = ic * hcol * wcol;
	dim3 blocks(GET_BLOCKS(op_size), bs, 1);
	dim3 threads(CUDA_NUM_THREADS, 1, 1);
	im2col_kernel<<<blocks, threads>>>(data_im, data_col, op_size, kh, kw, pad, stride, ih, iw, ic, hcol, wcol);
	//CUDA_POST_KERNEL_CHECK; // check if there was any error

	// now, the col form shall be multiplied with the kernels laid out straight i.e. (ic * kh * kw)
	// so, since, oc is the number of kernels, we get:
	// "2D kernel matrix" oc x (ic * kh * kw)
	// and the "2D col matrix" for one image is: (ic * kh * kw) x (hcol * wcol)
	// and you see that magically, their multiplication output is:
	// output: oc x (hcol * wcol)... ie oc x hcol x wcol, the exact shape needed by next convolution
	// output: oc x (hcol * wcol)... ie oc x hcol x wcol, the exact shape needed by next im2col
	// so, there is no need to ever work things back (col2im) or reshape either
	// in sumamary, we do matmul(kernel, im2col(im_input)) -> conv_output (in "correct" form)

	// Step 2: GEMM using libcublas

	// get params ready for GEMM call
	// Performs C + i*strideC = α op(A + i*strideA) op(B + i*strideB) + β(C + i* strideC) 
	// for i ∈ [0, batchSize − 1]
	// Thus, this one call will parallely do the matrix multiplication for all images in the batch
	// Since we are doing A * B, we need α = 1, β = 0
	// Since we don't need any transpose, op = HIPBLAS_OP_N
	const float alpha = 1.0f;
	const float beta  = 0.0f;
	int ldA, ldB, ldC;
	int m = ldA = ldC = hcol * wcol;
	int n = oc;
	int k = ldB = ic * kh * kw;
	long long int strideA = m * k;	// size of each col form
	long long int strideB = 0;		// reusing the same kernel matrix for each image
	long long int strideC = m * n;	// size of output feature map
	
	// CUDA sees matrices as column major
	// So, a matrix we see as HxW, it would see as WxH in the same memory layout
	// So, matA (our view) -> matA' (CUDA view)
	// Thus, to do matA * matB in our view, we shall run CUDA for matB * matA.
	// Output would be matB' * matA' (CUDA view) = (matA * matB)' (CUDA view) = matA * matB (our view)
	// In essence, trust me when I do col * kernel to achieve kernel * col

	// Blocks per grid dimension (assumes CUDA_NUM_THREADS divides M and n evenly)
	int BLOCKS_X = n / CUDA_NUM_THREADS;
	int BLOCKS_Y = m / CUDA_NUM_THREADS;
  
	// Use dim3 structs for block  and grid dimensions
	dim3 threads2(CUDA_NUM_THREADS, CUDA_NUM_THREADS);
	dim3 blocks2(BLOCKS_X, BLOCKS_Y);
  
	// Launch kernel
	matrixMul<<<blocks2, threads2>>>(data_col, data_ker, data_out, m, n, k, bs);
}


// takes a batch of images on CPU: data_im:  batch x ic x ih x iw (ic: input channels)
// and the kernels on CPU: data_ker: oc x ic x kh x kw (oc: output channels)
// does the convolution based on padding (pad) and stride
// returns the convolution output on CPU
// conv_time & overhead_time are used for kernel timing
float * im2colWithCuda(const float * data_im, const float * data_ker, const int batch,
					   const int kh, const int kw, const int pad, const int stride,
					   const int ih, const int iw, const int ic, const int oc, 
					   float& conv_time, float& overhead_time)
{
	
	// Timing variables - CUDA Event API
	overhead_time = 0;
	conv_time = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);  

	// image dim
	ssize_t image_size = ic * ih * iw;
	ssize_t images_size = batch * image_size;

	// kernel dim
	ssize_t k = ic * kh * kw;
	ssize_t kernels_size = oc * k;

	// col dim
	ssize_t hcol = (ih + 2 * pad - kh) / stride + 1;
	ssize_t wcol = (iw + 2 * pad - kw) / stride + 1;
	ssize_t one_col = ic * kh * kw * hcol * wcol;
	ssize_t col_batch = batch * one_col;

	// output dim
	ssize_t output_feature = oc * hcol * wcol;	
	ssize_t result_size = batch * output_feature;
	
	// move images to GPU
	float * dev_image = nullptr;
	CUDA_CHECK(hipMalloc((void**)&dev_image, images_size * sizeof(float)));
	CUDA_CHECK(hipMemcpy(dev_image, data_im, images_size * sizeof(float), hipMemcpyHostToDevice));
	
	// move kernels to GPU
	float * dev_kernel = nullptr;
	CUDA_CHECK(hipMalloc((void**)&dev_kernel, kernels_size * sizeof(float)));
	CUDA_CHECK(hipMemcpy(dev_kernel, data_ker, kernels_size * sizeof(float), hipMemcpyHostToDevice));

	// allocate GPU memory for intermediate col form
	float * dev_col = nullptr;
	CUDA_CHECK(hipMalloc((void**)&dev_col, col_batch * sizeof(float)));

	// allocate GPU memory for convlution result
	float * dev_ret = nullptr;
	CUDA_CHECK(hipMalloc((void**)&dev_ret, result_size * sizeof(float)));
	
	// Record the kernel run time
	hipEventRecord(start);
	// Kernel launch - this single call will handle all the images in the batch parallely
	im2col_gemm_gpu(dev_image, dev_kernel, kh, kw, pad, stride, ih, iw, ic, oc, dev_col, dev_ret, batch);
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	hipEventElapsedTime(&conv_time, start, stop);
	
	// Check for any errors launching the kernel
	//CUDA_POST_KERNEL_CHECK;

	// Copy output vector from GPU to host memory.
	float * data_ret = (float *)malloc(result_size * sizeof(float));
	CUDA_CHECK(hipMemcpy(data_ret, dev_ret, result_size * sizeof(float), hipMemcpyDeviceToHost));

	// Free CUDA memory
	hipFree(dev_image);
	hipFree(dev_col);
	hipFree(dev_kernel);
	hipFree(dev_ret);
	
	// Free timing resources
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return data_ret;
}

// The exposed library function which just calls im2colWithCuda the right way
float* IM2COL::forward(int out_size, int channel, int kernel_height, int kernel_width, int pad, 
		int stride, float* kernel, int batch_size, int input_height, int input_width, float* input, 
		float& conv_time, float& overhead_time)
{
	return im2colWithCuda(input, kernel, batch_size, kernel_height, kernel_width, 
					pad, stride, input_height, input_width, channel, out_size, conv_time, overhead_time);
}
